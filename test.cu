#include <iostream>
#include <chrono>
#include <random>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void inv(float *a, float *b, int n){

    hipblasHandle_t handle; 
    hipblasCreate(&handle);

    float *A[1] = {a};
    float *B[1] = {b};
    int *info;
    int *p;

    hipMalloc(&info, sizeof(int));
    hipMalloc(&p, n * sizeof(int));
    hipblasSgetrfBatched(handle, n, A, n, p, info, 1);
    hipblasSgetriBatched(handle, n, A, n, p, B, n, info, 1);
    hipFree(info);
    hipFree(p);
    hipblasDestroy(handle);
}

void dot(float *a, float *b, float *y, int n){
    for(int i = 0; i < n; ++i)
    {
        for(int j = 0; j < n; ++j)
        {
            double r = 0;
            for(int k = 0; k < n; ++k)
            {
                r += a[i*n+k] * b[k*n+j];
            }
            y[i*n+j] = abs(r)<0.001?0: r;
        }
    }
}

int main(){
    int n = 100;
    // std::cin>>n;

    std::chrono::system_clock::time_point  start, end;
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    float *ref = (float*)calloc(n*n, sizeof(float));
    for(int i = 0;i<n*n;i++){
        ref[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }
    std::cout<<std::endl;

    float *a, *b;
    // *a = (float*)calloc(n*n, sizeof(float));
    // *b = (float*)calloc(n*n, sizeof(float));
    hipMalloc(&a, n*n*sizeof(float));
    hipMalloc(&b, n*n*sizeof(float));
    float *buf = (float*)calloc(n*n, sizeof(float));
    float *a_ = (float*)calloc(n*n, sizeof(float));
    float *b_ = (float*)calloc(n*n, sizeof(float));
    float *y_ = (float*)calloc(n*n, sizeof(float));
    // cudaMallocManaged(&a, n*n*sizeof(float));
    // cudaMallocManaged(&b, n*n*sizeof(float));
    for(int i = 0;i<n*n;i++){
        a_[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }
    // for(int i = 0;i<n;i++){
    //     a__[i] = MyRand(mt);
    // }

    // float *a;
    // CHECK(cudaMallocManaged(&a, n*n*sizeof(float)));
    // float *b;
    // CHECK(cudaMallocManaged(&b, n*n*sizeof(float)));




    // for(int j = 10;j<n;j*=10){
    //     for(int i=1;i<10;++i){
    //         int N = j*i;
    //         cudaMemcpy(ref, a, n*n*sizeof(float), cudaMemcpyHostToDevice);
    //         start = std::chrono::system_clock::now();
    //         inv(a, b, N);
    //         end = std::chrono::system_clock::now();


    //         double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    //         std::cout<<N<<"\t"<<elapsed<<std::endl;


    //     }

    // }

    double sum=0;
    hipMemcpy(a, a_, n*n*sizeof(float), hipMemcpyHostToDevice);
    
    for(int i = 0;i<n*n;++i)sum+=a_[i];
    std::cout<<sum<<std::endl;
    // showArray(a_, n);
    inv(a, b, n);
    hipDeviceSynchronize();
    hipMemcpy(buf, a, n*n*sizeof(float), hipMemcpyDeviceToHost);
    // showArray(buf, n);
    hipMemcpy(b_, b, n*n*sizeof(float), hipMemcpyDeviceToHost);
    // showArray(b_, n);
    std::cout<<"meow"<<std::endl;
    dot(a_, b_, y_, n);
    sum = 0;
    for(int i = 0;i<n*n;++i)sum+=buf[i];
    std::cout<<sum<<std::endl;
    sum = 0;
    for(int i = 0;i<n*n;++i)sum+=y_[i];
    std::cout<<sum<<std::endl;
    // showArray(y_, n);




    return 0;
}