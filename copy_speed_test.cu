#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>

__global__ void mkarry(float *a){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    a[index] = 0;
}


int main(){
    float *a;
    int N = 8*32*1000*1000*sizeof(float);
    float milliseconds;
    hipMalloc(&a, N);
    hipMemset(&a, 0, N);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    milliseconds = 0;
    hipEventRecord(start);
    hipMemset(&a, 0, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<N<<"\t"<<milliseconds<<std::endl;

    milliseconds = 0;
    hipEventRecord(start);
    mkarry<<<N/32, 32>>>(a);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout<<N<<"\t"<<milliseconds<<std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);

}