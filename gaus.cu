
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

void inputArray(float *a, int n){
    for(int i=0;i<n*n;++i){
        std::cin >> a[i];
    }
}

void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}


__global__ void mkide(float *a, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    a[index*n+index] = 1;
}

__global__ void divRow(float *a, float *b, float *s, int n, int i){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // printf("%d\n", index);
    if(index<n){
        // printf("%f\n", b[i*n+index]);
        float t = a[i*n+i];
        a[i*n+index] /= t;
        b[i*n+index] /= t;
        s[index] = a[index * n + i];
        // printf("%f %f\n", b[i*n+index], a[i*n+i]);
    }
}

// __global__ void GaussElimination(float *a, float *b, int n, int i){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index != i && index < n){
//         float t = a[index*n+i];
//         for(int k = 0; k < n; ++k){
//             a[index*n+k] -= a[i*n+k]*t;
//             b[index*n+k] -= b[i*n+k]*t;
//         }
//     }
// }

__global__ void GaussElimination(float *a, float *b, float *t, int n, int i){
    //-- a?b[i*n:i*n+k]はキャッシュorシェアード --、tはコンスタントメモリを使うべき?
    //無駄になるスレッド多いし要修正
    int col = blockIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.x;
    // printf("%d %d\n", col, row);
    // if(col >= n || row >= n || i == col || i == row) return;
    if(row >= n || i == col) return;
    int index = col * n + row;
    // printf("%d %f %f\n", index, a[index], t[col]);

    // a[index*n+k] -= __ldg(a[i*n+k])*t[col];
    // b[index*n+k] -= __ldg(b[i*n+k])*t[col];
    a[index] -= a[i*n+row]*t[col];
    b[index] -= b[i*n+row]*t[col];
}

__constant__ float t[10000];
__host__ void GaussJordanGpuOptimize(float *a, float *b, int n){
    int blockSize = n/32 + (n%32 ? 1 : 0);
    mkide<<<blockSize,32>>>(b, n);
    hipDeviceSynchronize();
    // for(int i = 0; i < n; ++i){
    //     b[i*n+i] = 1;
    // }
    dim3 thread(32);
    dim3 block(n, n/32 + n%32!=0);

    // printf("%d, %d, %d\n", n, n/32 + n%32!=0, 32);
    float *s;
    hipMalloc(&s, sizeof(float)*n);

    for(int i = 0;i<n; ++i){
        int in = i*n;
        // std::cout<<i<<" "<<in<<" "<<&a[in]<<" "<<&b[in]<<std::endl;
        // std::cout<<thread.x<<" "<<thread.y<<" "<<thread.z<<std::endl;
        // std::cout<<block.x<<" "<<block.y<<" "<<block.z<<std::endl;
        divRow<<<blockSize, 32>>>(a, b, s, n, i);
        hipDeviceSynchronize();
        // cudaMemcpyToSymbol(t, &a[in], n, cudaMemcpyDeviceToDevice);
        GaussElimination<<<block, thread>>>(a, b, s, n, i);
        hipDeviceSynchronize();
    }
}

// __global__ void GaussJordanGpuOptimize(float *a, float *b, int n){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index>=n)return;
//     a[index*n+index] = 1;
//     __syncthreads(); 
//     for(int i = 0;i<n; ++i){
//         int in = i*n;
//         float t = a[in+i];
//         a[in+index] /= t;
//         b[in+index] /= t;
//         __syncthreads();
//         for(int j=0;j<n;++j){
//             if(j != i){
//                 float t = a[j*n+i];
//                 a[j*n+index] -= a[in+index]*t;
//                 b[j*n+index] -= b[in+index]*t;
//             }
//         }
//         __syncthreads();
//     }
// }

__global__ void GaussJordanGpu(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}

void GaussJordan(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}


int main(){
    int n = 10000;
    // std::cin>>n;
    std::chrono::system_clock::time_point  start, end; 
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    float *ref = (float*)calloc(n*n, sizeof(float));
    for(int i = 0;i<n*n;i++){
        ref[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }
    std::cout<<std::endl;
    
    float *a, *b, *buf;
    // *a = (float*)calloc(n*n, sizeof(float));
    // *b = (float*)calloc(n*n, sizeof(float));
    hipMalloc(&a, n*n*sizeof(float));
    hipMalloc(&b, n*n*sizeof(float));
    buf = (float*)calloc(n*n, sizeof(float));
    // cudaMallocManaged(&a, n*n*sizeof(float));
    // cudaMallocManaged(&b, n*n*sizeof(float));
    for(int i = 0;i<n*n;i++){
        buf[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }

    
    // float *a;
    // CHECK(cudaMallocManaged(&a, n*n*sizeof(float)));
    // float *b;
    // CHECK(cudaMallocManaged(&b, n*n*sizeof(float)));

    
    
    
    for(int j = 10;j<=10000;j*=10){
        for(int i=1;i<10;++i){
            int N = j*i;
            int blockSize = N/32 + (N%32 ? 1 : 0);
            hipMemcpy(ref, a, n*n*sizeof(float), hipMemcpyHostToDevice);
            start = std::chrono::system_clock::now();
            // GaussJordan(a, b, N);
            GaussJordanGpuOptimize(a, b, N);
            // GaussJordanGpuOptimize<<<blockSize, 32>>>(a, b, N);
            hipDeviceSynchronize();
            
            end = std::chrono::system_clock::now();
            
            
            double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
            std::cout<<N<<"\t"<<elapsed<<std::endl;
            
            
        }
        
    }
    // std::cout<<"hoge"<<std::endl;
    // inputArray(a, n);
    // std::cout<<"hoge"<<std::endl;
    // cudaMemcpy(a, buf, n*n*sizeof(float), cudaMemcpyHostToDevice);
    // showArray(buf, n);
    // GaussJordanGpuOptimize(a, b, n);
    // cudaDeviceSynchronize();
    // cudaMemcpy(buf, a, n*n*sizeof(float), cudaMemcpyDeviceToHost);
    // showArray(buf, n);
    // cudaMemcpy(buf, b, n*n*sizeof(float), cudaMemcpyDeviceToHost);
    // showArray(buf, n);


    return 0;
}