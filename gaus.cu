
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

void inputArray(float *a, int n){
    for(int i=0;i<n*n;++i){
        std::cin >> a[i];
    }
}

void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}


__global__ void mkide(float *a, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    a[index*n+index] = 1;
}

__global__ void divRow(float *a, float *b, float t, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", index);
    if(index<n){
        a[index] /= t;
        b[index] /= t;
    }
}

// __global__ void GaussElimination(float *a, float *b, int n, int i){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index != i && index < n){
//         float t = a[index*n+i];
//         for(int k = 0; k < n; ++k){
//             a[index*n+k] -= a[i*n+k]*t;
//             b[index*n+k] -= b[i*n+k]*t;
//         }
//     }
// }

__global__ void GaussElimination(float *a, float *b, float *t, int n, int i){
    //-- a?b[i*n:i*n+k]はキャッシュorシェアード --、tはコンスタントメモリを使うべき?
    //無駄になるスレッド多いし要修正
    int col = blockIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.x;
    if(col >= n || row >= n || i == col || i == row) return;
    int index = col * n + row;

    printf("%d %d\n", col, row);
    // a[index*n+k] -= __ldg(a[i*n+k])*t[col];
    // b[index*n+k] -= __ldg(b[i*n+k])*t[col];
    a[index*n+row] -= a[i*n+row]*t[col];
    b[index*n+row] -= b[i*n+row]*t[col];
}

__constant__ float t[10000];
__host__ void GaussJordanGpuOptimize(float *a, float *b, int n){
    int blockSize = n/32 + (n%32 ? 1 : 0);
    mkide<<<blockSize,32>>>(b, n);
    hipDeviceSynchronize();
    // for(int i = 0; i < n; ++i){
    //     b[i*n+i] = 1;
    // }
    dim3 thread(32);
    dim3 block(n, n/32 + n%32!=0);

    printf("%d, %d, %d\n", n, n/32 + n%32!=0, 32);
    // float *t_;
    // cudaMalloc(&t_, sizeof(float)*n);
    for(int i = 0;i<n; ++i){
        int in = i*n;
        divRow<<<blockSize, 32>>>(&a[in], &b[in], a[in+i], n);
        hipDeviceSynchronize();
        hipMemcpyToSymbol(HIP_SYMBOL(t), a, n, in, hipMemcpyDeviceToDevice);
        GaussElimination<<<block, thread>>>(a, b, t, n, i);
        hipDeviceSynchronize();
        std::cout<<i<<" "<<in<<std::endl;
        std::cout<<thread.x<<" "<<thread.y<<" "<<thread.z<<std::endl;
        std::cout<<block.x<<" "<<block.y<<" "<<block.z<<std::endl;
    }
}

// __global__ void GaussJordanGpuOptimize(float *a, float *b, int n){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index>=n)return;
//     a[index*n+index] = 1;
//     __syncthreads(); 
//     for(int i = 0;i<n; ++i){
//         int in = i*n;
//         float t = a[in+i];
//         a[in+index] /= t;
//         b[in+index] /= t;
//         __syncthreads();
//         for(int j=0;j<n;++j){
//             if(j != i){
//                 float t = a[j*n+i];
//                 a[j*n+index] -= a[in+index]*t;
//                 b[j*n+index] -= b[in+index]*t;
//             }
//         }
//         __syncthreads();
//     }
// }

__global__ void GaussJordanGpu(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}

void GaussJordan(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}


int main(){
    int n = 10;
    // std::cin>>n;
    std::chrono::system_clock::time_point  start, end; 
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    float *ref = (float*)calloc(n*n, sizeof(float));
    for(int i = 0;i<n*n;i++){
        ref[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }
    std::cout<<std::endl;
    
    float *a, *b;
    // *a = (float*)calloc(n*n, sizeof(float));
    // *b = (float*)calloc(n*n, sizeof(float));
    // cudaMalloc(&a, n*n*sizeof(float));
    // cudaMalloc(&b, n*n*sizeof(float));
    hipMallocManaged(&a, n*n*sizeof(float));
    hipMallocManaged(&b, n*n*sizeof(float));
    for(int i = 0;i<n*n;i++){
        a[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }

    
    // float *a;
    // CHECK(cudaMallocManaged(&a, n*n*sizeof(float)));
    // float *b;
    // CHECK(cudaMallocManaged(&b, n*n*sizeof(float)));

    
    
    
    // for(int j = 10;j<10000;j*=10){
    //     for(int i=1;i<9;++i){
    //         int N = j*i;
    //         int blockSize = N/32 + (N%32 ? 1 : 0);
    //         cudaMemcpy(ref, a, n*n*sizeof(float), cudaMemcpyHostToDevice);
    //         start = std::chrono::system_clock::now();
    //         // GaussJordan(a, b, N);
    //         GaussJordanGpuOptimize(a, b, N);
    //         // GaussJordanGpuOptimize<<<blockSize, 32>>>(a, b, N);
    //         cudaDeviceSynchronize();
            
    //         end = std::chrono::system_clock::now();
            
            
    //         double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    //         std::cout<<N<<"\t"<<elapsed<<std::endl;
            
            
    //     }
        
    // }
    // std::cout<<"hoge"<<std::endl;
    // inputArray(a, n);
    // std::cout<<"hoge"<<std::endl;
    // cudaMemcpy(ref, a, n*n*sizeof(float), cudaMemcpyHostToDevice);
    showArray(a, n);
    GaussJordanGpuOptimize(a, b, n);
    hipDeviceSynchronize();
    showArray(a, n);
    showArray(b, n);


    return 0;
}