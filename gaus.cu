
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <random>
#include <math.h>


#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
    }                                                                          \
}

void dot(float *a, float *b, float *y, int n){
    for(int i = 0; i < n; ++i)
    {
        for(int j = 0; j < n; ++j)
        {
            double r = 0;
            for(int k = 0; k < n; ++k)
            {
                r += a[i*n+k] * b[k*n+j];
            }
            y[i*n+j] = abs(r)<0.001?0: r;
        }
    }
}


void inputArray(float *a, int n){
    for(int i=0;i<n*n;++i){
        std::cin >> a[i];
    }
}

void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}


__global__ void mkide(float *a, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    a[index*n+index] = 1;
}

__global__ void divRow(float *a, float *b, float *s, int n, int i){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index<n){
        // printf("%d\n", index);
        // printf("%f\n", b[i*n+index]);
        float t = a[i*n+i];
        a[i*n+index] /= t;
        b[i*n+index] /= t;
        s[index] = a[index * n + i];
        // printf("%f %f\n", b[i*n+index], a[i*n+i]);
    }
}

// __global__ void GaussElimination(float *a, float *b, int n, int i){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index != i && index < n){
//         float t = a[index*n+i];
//         for(int k = 0; k < n; ++k){
//             a[index*n+k] -= a[i*n+k]*t;
//             b[index*n+k] -= b[i*n+k]*t;
//         }
//     }
// }

__global__ void GaussElimination(float *a, float *b, float *t, int n, int i){
    //-- a?b[i*n:i*n+k]はキャッシュorシェアード --、tはコンスタントメモリを使うべき?
    //無駄になるスレッド多いし要修正
    int col = blockIdx.x;
    int row = blockIdx.y*blockDim.x + threadIdx.x;
    // if(col >= n || row >= n || i == col || i == row) return;
    if(row >= n || i == col) return;
    // printf("%d %d %d\n", col, row, blockDim.x);
    int index = col * n + row;
    // printf("%d %f %f\n", index, a[index], t[col]);

    // a[index*n+k] -= __ldg(a[i*n+k])*t[col];
    // b[index*n+k] -= __ldg(b[i*n+k])*t[col];
    a[index] -= a[i*n+row]*t[col];
    b[index] -= b[i*n+row]*t[col];
}

__constant__ float t[10000];
__host__ void GaussJordanGpuOptimize(float *a, float *b, int n){
    
    int blockSize = n/32 + (n%32 ? 1 : 0);
    printf("blockSize %d\n",blockSize);
    mkide<<<blockSize,32>>>(b, n);
    hipDeviceSynchronize();
    // for(int i = 0; i < n; ++i){
    //     b[i*n+i] = 1;
    // }
    dim3 thread(32);
    dim3 block(n, n/32 + (n%32 ? 1 : 0));

    printf("%d, %d, %d\n", n, n/32 + (n%32 ? 1 : 0), 32);
    float *s;
    hipMalloc(&s, sizeof(float)*n);

    for(int i = 0;i<n; ++i){
        int in = i*n;
        // std::cout<<i<<" "<<in<<" "<<&a[in]<<" "<<&b[in]<<std::endl;
        // std::cout<<thread.x<<" "<<thread.y<<" "<<thread.z<<std::endl;
        // std::cout<<block.x<<" "<<block.y<<" "<<block.z<<std::endl;
        divRow<<<blockSize, thread>>>(a, b, s, n, i);
        hipDeviceSynchronize();
        // cudaMemcpyToSymbol(t, &a[in], n, cudaMemcpyDeviceToDevice);

        GaussElimination<<<block, thread>>>(a, b, s, n, i);
        CHECK(hipDeviceSynchronize());

    }
}

// __global__ void GaussJordanGpuOptimize(float *a, float *b, int n){
//     int index = blockIdx.x * blockDim.x + threadIdx.x;
//     if(index>=n)return;
//     a[index*n+index] = 1;
//     __syncthreads(); 
//     for(int i = 0;i<n; ++i){
//         int in = i*n;
//         float t = a[in+i];
//         a[in+index] /= t;
//         b[in+index] /= t;
//         __syncthreads();
//         for(int j=0;j<n;++j){
//             if(j != i){
//                 float t = a[j*n+i];
//                 a[j*n+index] -= a[in+index]*t;
//                 b[j*n+index] -= b[in+index]*t;
//             }
//         }
//         __syncthreads();
//     }
// }

__global__ void GaussJordanGpu(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}

void GaussJordan(float *a, float *b, int n){
    for(int i = 0; i < n; ++i){
        b[i*n+i] = 1;
    }

    for(int i = 0; i < n; ++i){
        float t = a[i*n+i];
        for(int j = 0; j < n; ++j){
            a[i*n+j] /= t;
            b[i*n+j] /= t;
        }
        for(int j = 0; j < n; ++j){
            if(i != j){
                float t = a[j*n+i];
                for(int k = 0; k < n; ++k){
                    a[j*n+k] -= a[i*n+k]*t;
                    b[j*n+k] -= b[i*n+k]*t;
                }
            }
        }
    }
}


int main(){
    int n = 1000;
    // std::cin>>n;
    std::chrono::system_clock::time_point  start, end; 
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    float *ref = (float*)calloc(n*n, sizeof(float));
    for(int i = 0;i<n*n;i++){
        ref[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }
    std::cout<<std::endl;
    
    float *a, *b;
    // *a = (float*)calloc(n*n, sizeof(float));
    // *b = (float*)calloc(n*n, sizeof(float));
    hipMalloc(&a, n*n*sizeof(float));
    hipMalloc(&b, n*n*sizeof(float));
    float *buf = (float*)calloc(n*n, sizeof(float));
    float *a_ = (float*)calloc(n*n, sizeof(float));
    float *b_ = (float*)calloc(n*n, sizeof(float));
    float *y_ = (float*)calloc(n*n, sizeof(float));
    // cudaMallocManaged(&a, n*n*sizeof(float));
    // cudaMallocManaged(&b, n*n*sizeof(float));
    for(int i = 0;i<n*n;i++){
        a_[i] = MyRand(mt);
        // std::cout<<ref[i]<<" ";
    }

    
    // float *a;
    // CHECK(cudaMallocManaged(&a, n*n*sizeof(float)));
    // float *b;
    // CHECK(cudaMallocManaged(&b, n*n*sizeof(float)));


    
    
    // for(int j = 10;j<=10000;j*=10){
    //     for(int i=1;i<10;++i){
    //         int N = j*i;
    //         int blockSize = N/32 + (N%32 ? 1 : 0);
    //         cudaMemcpy(ref, a, n*n*sizeof(float), cudaMemcpyHostToDevice);
    //         start = std::chrono::system_clock::now();
    //         // GaussJordan(a, b, N);
    //         GaussJordanGpuOptimize(a, b, N);
    //         // GaussJordanGpuOptimize<<<blockSize, 32>>>(a, b, N);
    //         cudaDeviceSynchronize();
            
    //         end = std::chrono::system_clock::now();
            
            
    //         double elapsed = std::chrono::duration_cast<std::chrono::milliseconds>(end-start).count();
    //         std::cout<<N<<"\t"<<elapsed<<std::endl;
            
            
    //     }
        
    // }

    // float *s;
    // cudaMalloc(&s, sizeof(float)*n);
    // dim3 thread(32);
    // dim3 block(n, n/32 + (n%32 ? 1 : 0));
    // cudaMemcpy(a, a_, n*n*sizeof(float), cudaMemcpyHostToDevice);
    // GaussElimination<<<block, thread>>>(a, b, s, n, i);



    // // std::cout<<"hoge"<<std::endl;
    // // inputArray(a, n);
    // // std::cout<<"hoge"<<std::endl;
    hipMemcpy(a, a_, n*n*sizeof(float), hipMemcpyHostToDevice);
    // showArray(a_, n);
    GaussJordanGpuOptimize(a, b, n);
    hipDeviceSynchronize();
    hipMemcpy(buf, a, n*n*sizeof(float), hipMemcpyDeviceToHost);
    // showArray(buf, n);
    hipMemcpy(b_, b, n*n*sizeof(float), hipMemcpyDeviceToHost);
    // showArray(b_, n);
    std::cout<<"meow"<<std::endl;
    dot(a_, b_, y_, n);
    double sum=0;
    for(int i = 0;i<n*n;++i)sum+=buf[i];
    std::cout<<sum<<std::endl;
    sum = 0;
    for(int i = 0;i<n*n;++i)sum+=y_[i];
    std::cout<<sum<<std::endl;
    // showArray(y_, n);



    return 0;
}