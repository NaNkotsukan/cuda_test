#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>



void inv(float *a, float *b, int n){
    hipsolverStatus_t status;
    hipsolverHandle_t handle;
    status = hipsolverDnCreate(&handle);
    float* A;
    hipMalloc(&A, sizeof(float)*n*n);
    hipMemcpy(A, A, sizeof(float)*n*n, hipMemcpyHostToDevice);
    int worksize;
    status = hipsolverDnSpotrf_bufferSize(handle, HIPBLAS_FILL_MODE_LOWER, 
        n,
        A,
        n,
        &worksize);
    std::cout << "worksize:" << worksize << std::endl;
    std::cout << "status:" << status << std::endl;
    float *workspace;
    hipMalloc(&workspace, sizeof(float)*worksize);
    int *devInfo;
    hipMalloc(&devInfo, sizeof(int));
    status = hipsolverDnSpotrf(handle,
           HIPBLAS_FILL_MODE_LOWER,
           n,
           A,
           n,
           workspace,
           worksize,
           devInfo);
    std::cout << "status:" << status << std::endl;

           
    status = hipsolverDnSpotri_bufferSize(handle,
                 HIPBLAS_FILL_MODE_LOWER,
                 n,
                 A,
                 n,
                 worksize);
    std::cout << "worksize:" << worksize << std::endl;
    std::cout << "status:" << status << std::endl;
                 
    status = hipsolverDnSpotri(handle,
            HIPBLAS_FILL_MODE_LOWER,
            n,
            A,
            n,
            workspace,
            worksize,
            devInfo);
    std::cout << "status:" << status << std::endl;
    hipMemcpy(b, A, sizeof(float)*n*n, hipMemcpyDeviceToHost);
}
                
int main(){
    int N = 1000;
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    
    float *A = (float*)calloc(N*N, sizeof(float));
    float *B = (float*)calloc(N*N, sizeof(float));
    for(int i = 0;i<N*N;i++){
        A[i] = MyRand(mt);
    }

    for(int j = 10;j<N;j*=10){
        for(int i=1;i<10;++i){
            int n = j*i;
            if(n>N) break;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            inv(A, B, n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            std::cout<<n<<"\t"<<milliseconds<<std::endl;
        }
    }


    return 0;
}