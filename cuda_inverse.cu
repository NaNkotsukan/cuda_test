#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>


void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}

__global__ void eye(float *a, int n){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index>=n)return;
    for(int i = 0;i<n;++i){
        a[i*n+index] = i==index ? 1 : 0;
    }
}

inline void mkide(float *a, int n){
    eye<<<n/32 + (n%32 ? 1 : 0), 32>>>(a, n);
}

void inv(float *a, float *b, int n){
    hipsolverStatus_t status, status0, status1, status2;
    hipsolverHandle_t handle;
    status = hipsolverDnCreate(&handle);
    float* A;
    float* B;
    hipMalloc(&B, sizeof(float)*n*n);
    mkide(B, n);
    hipMalloc(&A, sizeof(float)*n*n);
    hipMemcpy(A, a, sizeof(float)*n*n, hipMemcpyHostToDevice);
    int worksize;
    float *workspace;
    int *devInfo;
    int *devIpiv;
    hipMalloc(&devInfo, sizeof(int));
    hipMalloc(&devIpiv, sizeof(int)*n);

    status0 = hipsolverDnSgetrf_bufferSize(handle,
        n, n,
        A,
        n,
        &worksize);
    hipMalloc(&workspace, sizeof(float)*worksize);
    // std::cout << "worksize:" << worksize << std::endl;
    // std::cout << "status:" << status << std::endl;
    
    status1 = hipsolverDnSgetrf(handle,
        n, n,
        A,
        n,
        workspace,
        devIpiv,
        devInfo);
    // std::cout << "status:" << status << std::endl;
    hipDeviceSynchronize();

    status2 = hipsolverDnSgetrs(handle,
        HIPBLAS_OP_N,
        n,
        n,
        A,
        n,
        devIpiv,
        B,
        n,
        devInfo);
    // std::cout << "status:" << status << status0 << status1 << status2 << std::endl;
    // std::cout << CUSOLVER_STATUS_SUCCESS<<" "<< CUSOLVER_STATUS_NOT_INITIALIZED<<" "<<CUSOLVER_STATUS_INVALID_VALUE <<" "<<CUSOLVER_STATUS_ARCH_MISMATCH<<" "<<CUSOLVER_STATUS_EXECUTION_FAILED<<" " <<CUSOLVER_STATUS_INTERNAL_ERROR << std::endl;
 
    hipMemcpy(b, B, sizeof(float)*n*n, hipMemcpyDeviceToHost);
}
                
int main(){
    int N = 10000;
    std::mt19937 mt(982359349);
    std::uniform_real_distribution<> MyRand(-1.0, 1.0);
    
    float *A = (float*)calloc(N*N, sizeof(float));
    float *B = (float*)calloc(N*N, sizeof(float));
    for(int i = 0;i<N*N;i++){
        A[i] = MyRand(mt);
    }

    for(int j = 1;j<N;j*=10){
        for(int i=1;i<10;++i){
            int n = j*i;
            if(n>N) break;
            hipEvent_t start, stop;
            hipEventCreate(&start);
            hipEventCreate(&stop);
            hipEventRecord(start);
            inv(A, B, n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);
            float milliseconds = 0;
            hipEventElapsedTime(&milliseconds, start, stop);
            hipEventDestroy(start);
            hipEventDestroy(stop);
            std::cout<<n<<"\t"<<milliseconds<<std::endl;
        }
    }

    // N = 10;
    // showArray(A, N);
    // inv(A, B, N);
    // showArray(B, N);

    return 0;
}