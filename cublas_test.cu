#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>


void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}

int main(){
    hipblasHandle_t cublas_handle;
    int n_samples, n_features, n_targets;
    n_samples = 5;
    n_features = 10;
    n_targets = 15;
    float *W;
    float *X, *Y;
    hipMallocManaged(&X, n_samples*n_features*sizeof(float));
    hipMallocManaged(&Y, n_samples*n_targets*sizeof(float));
    hipMallocManaged(&W, n_features*n_targets*sizeof(float));
    float *coef_matrix;
    hipMallocManaged(&coef_matrix, n_features * n_features * sizeof(float));
    for(int i = 0;i<n_samples*n_features;++i){
        X[i] = i;
        // Y[i] = i;
    }
    for(int i = 0;i<n_features*n_targets;++i){
        W[i] = i;
        // Y[i] = i;
    }

    for(int i = 0;i < n_features * n_features;++i){
        coef_matrix[i]= 0;
    }
    float rho = 1.0;            

    float inv_n_samp = 1.0f/n_samples;
    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features, n_features, n_samples,
        1.0f/n_samples,
        X, n_samples,
        W, n_samples,
        rho,
        Y,
        n_features);
    hipDeviceSynchronize();
    showArray(Y, n_features);

    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
        n_features, n_features, n_samples,
        1.0f/n_samples,
        X, n_samples,
        W, n_samples,
        rho,
        Y,
        n_features);

    hipDeviceSynchronize();
    showArray(Y, n_features);
    

    return 0;
}

