#include <iostream>
#include <random>
#include <hipsolver.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>


void showArray(float *a, int n){
    for(int i=0;i<n;++i){
        for(int j=0;j<n;++j){
            std::cout<<a[i*n+j]<<" ";
        }
        std::cout<<std::endl;
    }
}

int main(){
    hipblasHandle_t cublas_handle;
    int n_samples, n_features, n_targets;
    hipblasCreate(&cublas_handle);
    n_samples = 5;
    n_features = 10;
    n_targets = 15;
    float *W;
    float *X, *Y;
    hipMallocManaged(&X, n_samples*n_features*sizeof(float));
    hipMallocManaged(&Y, n_samples*n_targets*sizeof(float));
    hipMallocManaged(&W, n_features*n_targets*sizeof(float));
    float *coef_matrix;
    hipMallocManaged(&coef_matrix, n_features * n_features * sizeof(float));
    for(int i = 0;i<n_samples*n_features;++i){
        X[i] = i;
        // Y[i] = i;
    }
    for(int i = 0;i<n_features*n_targets;++i){
        W[i] = i;
        // Y[i] = i;
    }

    for(int i = 0;i < n_features * n_features;++i){
        coef_matrix[i]= 0;
    }
    float rho = 0;            

    float inv_n_samp = 1.0f/n_samples;
    float alpha = 1.0f;
    hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n_samples, n_targets, n_features,
        &alpha,
        X, n_samples,
        W, n_features,
        &rho,
        Y,
        n_samples);
    hipDeviceSynchronize();
    showArray(Y, 15);

    hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
        n_targets, n_samples, n_features,
        &alpha,
        W, n_features,
        X, n_samples,
        &rho,
        Y,
        n_targets);
    hipDeviceSynchronize();
    showArray(Y, 15);

    // cublasSgemm(cublas_handle, CUBLAS_OP_N, CUBLAS_OP_T,
    //     n_features, n_features, n_samples,
    //     &alpha,
    //     X, n_samples,
    //     W, n_samples,
    //     &rho,
    //     Y,
    //     n_features);

    // cudaDeviceSynchronize();
    // showArray(Y, n_features);
    

    return 0;
}

